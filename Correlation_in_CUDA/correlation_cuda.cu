
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <chrono>
#include <iostream>
#include <omp.h>
// #include <fftw3.h>
#include <cmath>

#define NUM_THREADS 128

void ComputeNorms(float *events, float *norms, int n_events, int event_length, int paddedSize){
  float pot =0.0;
  for(int i=0; i<n_events; i++){
    pot=0.0;
    for(int j=0; j<event_length; j++){
      pot += events[i*paddedSize+j] * events[i*paddedSize+j];
    }
    norms[i] = sqrt(pot);
  }
}

__global__ void ElementWiseMultiplication(hipfftComplex *d_events_freq, hipfftComplex *d_events_reversed_freq,
                int n_events, int fftsize, int chunk_number, int chunkSize, hipfftComplex *d_corr_f){
  int tid = threadIdx.x;
  int i, j;
  //int block = blockIdx.x % chunkSize;
  i = blockIdx.y + chunk_number * chunkSize;
  j = blockIdx.x;

  //Create a loop where every thread computes several multiplications of the same xcorr
  for(int k=0; k<int(fftsize/blockDim.x); k++){
    d_corr_f[(blockIdx.y * n_events + blockIdx.x) * fftsize + tid + k * blockDim.x].x =
                             d_events_freq[i * fftsize + tid + k * blockDim.x].x
                           * d_events_reversed_freq[j * fftsize + tid + k * blockDim.x].x
                           - d_events_freq[i * fftsize + tid + k * blockDim.x].y
                           * d_events_reversed_freq[j * fftsize + tid + k * blockDim.x].y;

    d_corr_f[(blockIdx.y * n_events + blockIdx.x) * fftsize + tid + k * blockDim.x].y =
                             d_events_freq[i * fftsize + tid + k * blockDim.x].y
                           * d_events_reversed_freq[j * fftsize + tid + k * blockDim.x].x
                           + d_events_freq[i * fftsize + tid + k * blockDim.x].x
                           * d_events_reversed_freq[j * fftsize + tid + k * blockDim.x].y;
  }

  if (tid == 0){
    d_corr_f[(blockIdx.y * n_events + blockIdx.x + 1) * fftsize - 1].x =
                             d_events_freq[(i+1) * fftsize - 1].x
                           * d_events_reversed_freq[(j+1) * fftsize - 1].x
                           - d_events_freq[(i+1) * fftsize - 1].y
                           * d_events_reversed_freq[(j+1) * fftsize -1].y;

    d_corr_f[(blockIdx.y * n_events + blockIdx.x + 1) * fftsize - 1].y =
                             d_events_freq[(i+1) * fftsize - 1].y
                           * d_events_reversed_freq[(j+1) * fftsize - 1].x
                           + d_events_freq[(i+1) * fftsize - 1].x
                           * d_events_reversed_freq[(j+1) * fftsize -1].y;
  }
}

__global__ void FeatureExtraction1TH(float *d_corr_t, int shift, int n_events, int paddedSize,
        int event_length, int chunk_number, int chunkSize, float *d_norms, float *d_xcorr_vals_pos,
        int *d_xcorr_lags_pos, float *d_xcorr_vals_neg, int *d_xcorr_lags_neg){

  int tid = threadIdx.x;

  if(tid == 0){
    int i, j;
    i = blockIdx.y + chunk_number * chunkSize;
    j = blockIdx.x;

    float val_pos = 0.0; float val_neg = 0.0;
    int lag_pos = 0;     int lag_neg = 0;

    for(int k = -shift; k < shift; k++){
      if(d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k]>val_pos){
        val_pos = d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k];
        lag_pos = k + 1;
      }
      else if(d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k]<val_neg){
        val_neg = d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k];
        lag_neg = k + 1;
      }
    }
    val_pos /= ((float) paddedSize * d_norms[i] * d_norms[j]);
    val_neg /= ((float) paddedSize * d_norms[i] * d_norms[j]);

    d_xcorr_vals_pos[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = val_pos;
    d_xcorr_vals_neg[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = val_neg;
    d_xcorr_lags_pos[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = lag_pos;
    d_xcorr_lags_neg[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = lag_neg;
  }
}


__global__ void FeatureExtraction(float *d_corr_t, int shift, int n_events, int paddedSize,
        int event_length, int chunk_number, int chunkSize, float *d_norms, float *d_xcorr_vals_pos,
        int *d_xcorr_lags_pos, float *d_xcorr_vals_neg, int *d_xcorr_lags_neg){

  int tid = threadIdx.x;
  int i, j;
  // int block = blockIdx.x % chunkSize;
  i = blockIdx.y + chunk_number * chunkSize;
  j = blockIdx.x;

  __shared__ float shared_max_val[NUM_THREADS];
  __shared__ int shared_max_lag[NUM_THREADS];
  __shared__ float shared_min_val[NUM_THREADS];
  __shared__ int shared_min_lag[NUM_THREADS];

  shared_max_val[tid] = 0.0;
  shared_max_lag[tid] = 0;
  shared_min_val[tid] = 0.0;
  shared_min_lag[tid] = 0;

  for(int k = -int(shift/blockDim.x); k<int(shift/blockDim.x); k++){ //generalise for the number of threads per block
    if(d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k * blockDim.x + tid] > shared_max_val[tid]){
      shared_max_val[tid] = d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k * blockDim.x + tid];
      shared_max_lag[tid] = k * blockDim.x + tid + 1;
    }
    else if(d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k * blockDim.x + tid] < shared_min_val[tid]){
      shared_min_val[tid] = d_corr_t[(blockIdx.y * n_events + blockIdx.x) * paddedSize + event_length + k * blockDim.x + tid];
      shared_min_lag[tid] = k * blockDim.x + tid + 1;
    }
    __syncthreads();
  }

  for (int stride = blockDim.x/2; stride>0; stride>>=1){
    if(tid < stride){
      if(shared_max_val[tid] < shared_max_val[tid + stride]){
        shared_max_val[tid] = shared_max_val[tid + stride];
        shared_max_lag[tid] = shared_max_lag[tid + stride];
      }

      if(shared_min_val[tid] > shared_min_val[tid + stride]){
        shared_min_val[tid] = shared_min_val[tid + stride];
        shared_min_lag[tid] = shared_min_lag[tid + stride];
      }
    }
    __syncthreads();
  }

  if(tid == 0){
    d_xcorr_vals_pos[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = shared_max_val[tid] / (d_norms[i] * d_norms[j] * paddedSize);
    d_xcorr_vals_neg[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = shared_min_val[tid] / (d_norms[i] * d_norms[j] * paddedSize);
    d_xcorr_lags_pos[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = shared_max_lag[tid];
    d_xcorr_lags_neg[(blockIdx.y + chunk_number * chunkSize) * n_events + blockIdx.x] = shared_min_lag[tid];
  }
}

void FeatureExtractionCPU (float *corr_t, int shift, int event_length, int paddedSize,
        float &norm1, float &norm2, float &val_pos, int &lag_pos, float &val_neg, int &lag_neg){

  val_pos = 0.0; val_neg = 0.0;
  lag_pos = 0;   lag_neg = 0;

  for(int i = 0; i < shift; i++){
    if(corr_t[i]>val_pos){
      val_pos = corr_t[i];
      lag_pos = i + 1;
    }
    else if(corr_t[i]<val_neg){
      val_neg = corr_t[i];
      lag_neg = i + 1;
    }
  }

  for(int i = event_length - shift; i <= event_length; i++){
    if(corr_t[i] > val_pos){
      val_pos = corr_t[i];
      lag_pos = i - event_length + 1;
    }
    else if(corr_t[i]<val_neg){
      val_neg = corr_t[i];
      lag_neg = i - event_length + 1;
    }
  }
  val_pos /= ((float) paddedSize * norm1 * norm2);
  val_neg /= ((float) paddedSize * norm1 * norm2);

}



void MultiplicationAndIFFT (hipfftComplex *d_events_freq, hipfftComplex *d_events_reversed_freq,
      int n_events, int event_length, int paddedSize, int fftsize, int shift, int chunkSize,
      int n_elements, int num_threads, float *d_norms, hipfftComplex *d_corr_f, float *d_corr_t,
      float *d_xcorr_vals_pos, int *d_xcorr_lags_pos, float *d_xcorr_vals_neg, int *d_xcorr_lags_neg){

  int num_chunks = int(n_events / chunkSize);

  int n[1] = {paddedSize};
  int num_rows_ifft = 1; //12

  hipfftHandle planIFFT;
  hipfftPlanMany(&planIFFT, 1, n,
              NULL, 1, fftsize,
              NULL, 1, paddedSize,
              HIPFFT_C2R, num_rows_ifft*n_events);

  dim3 gridDim = {unsigned(n_events), unsigned(chunkSize), 1U}; //unsigned(chunkSize) unsigned(2)


  for(int k=0; k<num_chunks; k++){
    // printf("Hola! Esta es la iteración %d\n", k);
    // printf("Las dimensiones del grid son: (%d, %d)\n", n_events, chunkSize);

    // auto time1=std::chrono::high_resolution_clock::now();
    ElementWiseMultiplication<<<gridDim, 512>>> (d_events_freq, d_events_reversed_freq,
                      n_events, fftsize, k, chunkSize, d_corr_f);
    hipDeviceSynchronize();
    // auto time2=std::chrono::high_resolution_clock::now();
    // std::cout << "Tiempo Multiplication: " << std::chrono::duration<double>(time2-time1).count() << std::endl;


    // auto time3=std::chrono::high_resolution_clock::now();
    for (int q=0; q<chunkSize/num_rows_ifft; q++){
      hipfftExecC2R (planIFFT, &d_corr_f[q * num_rows_ifft * n_events * fftsize],
                              &d_corr_t[q * num_rows_ifft * n_events * paddedSize]);
    }
    // auto time4=std::chrono::high_resolution_clock::now();
    // std::cout << "Tiempo IFFT: " << std::chrono::duration<double>(time4-time3).count() << std::endl;


    // auto time5=std::chrono::high_resolution_clock::now();
    FeatureExtraction<<<gridDim, NUM_THREADS>>> (d_corr_t, shift, n_events, paddedSize, event_length,
                      k, chunkSize, d_norms, d_xcorr_vals_pos, d_xcorr_lags_pos, d_xcorr_vals_neg,
                      d_xcorr_lags_neg);
    hipDeviceSynchronize();
    // auto time6=std::chrono::high_resolution_clock::now();
    // std::cout << "Tiempo FeatureExtraction events: " << std::chrono::duration<double>(time6-time5).count() << std::endl;

  }
}

extern "C"{
  //Function used to initialise the CUDA RunTime
  void initialiseCUDA(){
    float *d_example;
    hipfftComplex *d_result;
    int number = 128;
    int fftsize = number/2 + 1;

    hipfftHandle planFFT;

    hipMalloc((void**) &d_example, sizeof(float) * number);
    hipMalloc((void**) &d_result, sizeof(hipfftComplex) * fftsize);

    int n[1] = {number};

    hipfftPlanMany(&planFFT, 1, n,
                NULL, 1, number,
                NULL, 1, fftsize,
                HIPFFT_R2C, 1);
    hipfftExecR2C(planFFT, d_example, d_result);
  }


  void correlationCUDA(float *events, float *events_reversed , int n_events, int event_length,
                      int shift, int paddedSize, int num_threads,
                      float *xcorr_vals_pos, int *xcorr_lags_pos,
                      float *xcorr_vals_neg, int *xcorr_lags_neg){
    // int nDevices;
    //
    // cudaGetDeviceCount(&nDevices);
    // for (int i = 0; i < nDevices; i++) {
    //   cudaDeviceProp prop;
    //   cudaGetDeviceProperties(&prop, i);
    //   printf("Device Number: %d\n", i);
    //   printf("  Device name: %s\n", prop.name);
    //   printf("  Memory Clock Rate (KHz): %d\n",
    //          prop.memoryClockRate);
    //   printf("  Memory Bus Width (bits): %d\n",
    //          prop.memoryBusWidth);
    //   printf("  Peak Memory Bandwidth (GB/s): %f\n",
    //          2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    //   printf("  Max Threads per Block: %d\n",
    //          prop.maxThreadsPerBlock);
    //   printf("  Max Threads Per MultiProcessor: %d\n",
    //          prop.maxThreadsPerMultiProcessor);
    //   printf("  Max Thread Dim: %d %d %d\n",
    //          prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    //   printf("  Max Grid Size: %d %d %d\n\n",
    //          prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    // }

    printf("\n------------------Values received from Python------------------\n");
    int fftsize = paddedSize / 2 + 1;
    printf("C: n_events: %d\n", n_events);
    printf("C: event_length: %d\n", event_length);
    printf("C: shift: %d\n", shift);
    printf("C: paddedSize: %d\n", paddedSize);
    printf("C: fftsize: %d\n", fftsize);
    printf("C: num_threads %d\n", num_threads);


    // Hardcoded variables
    int chunkSize = 1309440 / 2; //numero de correlaciones cruzadas que caben en la memoria global
    // entre dos porque tenemos la correlacion en tiempo y en frecuencia
    // chunkSize /= n_events;
    chunkSize = 8; //96 //12
    printf("C: El tamaño del chunk es: %d\n", chunkSize);
    int n_elements = int(n_events * (n_events+1) / 2);

    hipfftHandle planFFT;
    hipfftHandle planFFTreversed;
    //cufftHandle planIFFT;

    float *d_events, *d_events_reversed, *d_corr_t, *d_norms;
    hipfftComplex *d_events_freq, *d_events_reversed_freq, *d_corr_f;

    float *d_xcorr_vals_pos, *d_xcorr_vals_neg;
    int *d_xcorr_lags_pos, *d_xcorr_lags_neg;


    printf("El peso de un hipfftComplex es: %d bytes\n", sizeof(hipfftComplex));
    //Reserva de memoria para los eventos en time-domain y copia desde el host
    hipMalloc((void**) &d_events, sizeof(float) * n_events * paddedSize);
    hipMemcpy(d_events, events, sizeof(float) * n_events * paddedSize, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_events_reversed, sizeof(float) * n_events * paddedSize);
    hipMemcpy(d_events_reversed, events_reversed, sizeof(float) * n_events * paddedSize, hipMemcpyHostToDevice);


    // Reserva de memoria para las FFTs
    hipMalloc((void**) &d_events_freq, sizeof(hipfftComplex) * n_events * fftsize);
    hipMalloc((void**) &d_events_reversed_freq, sizeof(hipfftComplex) * n_events * fftsize);

    // Reserva de memoria para las correlaciones en frecuencia
    hipMalloc((void**) &d_corr_f, sizeof(hipfftComplex) * chunkSize * n_events * fftsize);
    hipMalloc((void**) &d_corr_t, sizeof(float) * chunkSize * n_events * paddedSize);

    // Reserva de memoria para las matrices de salida
    hipMalloc((void**) &d_xcorr_vals_pos, sizeof(float) * n_events * n_events);
    hipMalloc((void**) &d_xcorr_vals_neg, sizeof(float) * n_events * n_events);
    hipMalloc((void**) &d_xcorr_lags_pos, sizeof(int) * n_events * n_events);
    hipMalloc((void**) &d_xcorr_lags_neg, sizeof(int) * n_events * n_events);


    float norms[n_events];
    ComputeNorms(events, norms, n_events, event_length, paddedSize);
    hipMalloc((void**) &d_norms, sizeof(float) * n_events);
    hipMemcpy(d_norms, norms, sizeof(float) * n_events, hipMemcpyHostToDevice);

    // THE FFT COMPUTATION STARTS
    int n[1] = {paddedSize};
    // Cabecera de la funcion: cufftExecR2C(cufftHandle plan, cufftReal *idata, cufftComplex *odata);

    auto time1=std::chrono::high_resolution_clock::now();
    hipfftPlanMany(&planFFT, 1, n,
                NULL, 1, paddedSize,
                NULL, 1, fftsize,
                HIPFFT_R2C, n_events);

    hipfftExecR2C(planFFT, d_events, d_events_freq);

    auto time2=std::chrono::high_resolution_clock::now();
    std::cout << "Tiempo FFT events: " << std::chrono::duration<double>(time2-time1).count() << std::endl;
    // cudaMemcpy(events_freq, d_events_freq, sizeof(cufftComplex) * n_events * fftsize, cudaMemcpyDeviceToHost);

    auto time3 = std::chrono::high_resolution_clock::now();
    hipfftPlanMany(&planFFTreversed, 1, n,
                NULL, 1, paddedSize,
                NULL, 1, fftsize,
                HIPFFT_R2C, n_events);

    hipfftExecR2C(planFFTreversed, d_events_reversed, d_events_reversed_freq);

    auto time4 = std::chrono::high_resolution_clock::now();
    std::cout << "Tiempo FFT events_reversed: " << std::chrono::duration<double>(time4-time3).count() << std::endl;

    hipFree(d_events);
    hipFree(d_events_reversed);


    MultiplicationAndIFFT(d_events_freq, d_events_reversed_freq, n_events, event_length,
            paddedSize, fftsize, shift, chunkSize, n_elements, num_threads, d_norms, d_corr_f, d_corr_t,
            d_xcorr_vals_pos, d_xcorr_lags_pos, d_xcorr_vals_neg, d_xcorr_lags_neg);
    auto time7 = std::chrono::high_resolution_clock::now();

    printf("Copying memory starts:\n");
    hipMemcpy (xcorr_vals_pos, d_xcorr_vals_pos, sizeof(float) * n_events * n_events, hipMemcpyDeviceToHost);
    hipMemcpy (xcorr_vals_neg, d_xcorr_vals_neg, sizeof(float) * n_events * n_events, hipMemcpyDeviceToHost);
    hipMemcpy (xcorr_lags_pos, d_xcorr_lags_pos, sizeof(int) * n_events * n_events, hipMemcpyDeviceToHost);
    hipMemcpy (xcorr_lags_neg, d_xcorr_lags_neg, sizeof(int) * n_events * n_events, hipMemcpyDeviceToHost);

    auto time8 = std::chrono::high_resolution_clock::now();
    std::cout << "Tiempo copia memoria: " << std::chrono::duration<double>(time8-time7).count() << std::endl;
    //cudaMemcpy(events_reversed_freq, d_events_reversed_freq, sizeof(cufftComplex) * n_events * fftsize, cudaMemcpyDeviceToHost);


    hipFree(d_corr_t);
    hipFree(d_norms);
    hipFree(d_events_freq);
    hipFree(d_events_reversed_freq);
    hipFree(d_corr_f);
    hipFree(d_xcorr_vals_pos);
    hipFree(d_xcorr_vals_neg);
    hipFree(d_xcorr_lags_pos);
    hipFree(d_xcorr_lags_neg);


    printf("C: Finished computing in the GPU\n");



  }
}
